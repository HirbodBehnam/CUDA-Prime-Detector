#define MAX_TURN 1024 * 1024

#include "hip/hip_runtime.h"


#include <ctime>
#include <iostream>
#include <cstdio>
#include <stdio.h>
#include <stdlib.h>

__global__ void runTest(unsigned long long *base,unsigned long long *number, unsigned long long *res)
{
	unsigned long long index = 5 + (*base + threadIdx.x + blockIdx.x * blockDim.x) * 6;
	if (*number % index == 0) 
		*res = index;
	index += 2;
	if (*number % index == 0) 
		*res = index;
}

int main(int argc, char *argv[])
{
	if(argc != 2)
	{
		printf("CUDA Prime Detector v1.0.1");
		printf("By Hirbod Behnam");
		printf("Source: https://github.com/HirbodBehnam/CUDA-Prime-Detector");
		printf("Usage: ProgramName.exe NUMBER_TO_TEST");
		return 2;
	}
	clock_t start = clock();//Benchmark
	const unsigned long long Number = strtoull(argv[1],nullptr,10);
	//At first check small primes
	if(Number == 2 || Number == 3 || Number == 5 || Number == 7)
	{
		std::cout << Number << " is PRIME.";
		return 0;
	}
	if(Number % 2 == 0)
	{
		std::cout << Number << " is NOT prime. It can be divided by 2";
		return 0;
	}
	if(Number % 3 == 0)
	{
		std::cout << Number << " is NOT prime. It can be divided by 3";
		return 0;
	}
	//Now check if number is smaller than MAX TURN; If it is compute it on CPU
	const unsigned long long TO = (unsigned long long) sqrtl((long double) Number);
	if(Number <= MAX_TURN * 6)
	{
		for(unsigned long long i = 5;i<= TO;i+=4)
		{
			if(Number % i == 0)
			{
				double elapsedTime = static_cast<double>(clock() - start) / CLOCKS_PER_SEC;
				std::cout << Number << " is NOT prime. It can be divided by "<< i<<std::endl;
				std::cout << "Calculated in " <<elapsedTime;
				return 0;
			}
			i+=2;
			if(Number % i == 0)
			{
				double elapsedTime = static_cast<double>(clock() - start) / CLOCKS_PER_SEC;
				std::cout << Number << " is NOT prime. It can be divided by "<< i <<std::endl;
				std::cout << "Calculated in " <<elapsedTime;
				return 0;
			}
		}
		std::cout << Number << " is PRIME.";
		return 0;
	}
	//Number is big enough to compute on GPU
	unsigned long long res = 0,base = 0,to = TO;
	unsigned long long *d_number, *d_res,*d_base;
	int size = sizeof(unsigned long long);
	hipMalloc((void **)&d_number, size);
	hipMalloc((void **)&d_res, size);
	hipMalloc((void **)&d_base, size);
	hipMemcpy(d_number, &Number, size, hipMemcpyHostToDevice);
	while (to >= MAX_TURN)
	{
		hipMemcpy(d_base, &base, size, hipMemcpyHostToDevice);
		runTest <<<1024, 1024 >>> (d_base, d_number, d_res);
		hipMemcpy(&res, d_res, size, hipMemcpyDeviceToHost);
		if(res != 0)
			goto END;
		to -= MAX_TURN;
		base += MAX_TURN;
	}
	//Compute the rest
	while (base % 6 != 5)
		base++;
	for(;base <= TO;base += 4)
	{
		if(Number % base == 0)
		{
			res = base;
			goto END;
		}
		base+=2;
		if(Number % base == 0)
		{
			res = base;
			goto END;
		}
	}
END:
	double elapsedTime = static_cast<double>(clock() - start) / CLOCKS_PER_SEC;
	hipFree(d_base);
	hipFree(d_res);
	hipFree(d_number);
	if (res == 0 || res == 1) 
		std::cout << Number << " is PRIME." <<std::endl;
	else 
		std::cout << Number << " is NOT prime. It can be divided by "<< res <<std::endl;
	std::cout << "Calculated in " <<elapsedTime;
    return 0;
}
